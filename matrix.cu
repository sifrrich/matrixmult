#include "hip/hip_runtime.h"
#include <stdio.h>
#include "matrix.h"

/* 
*http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api 
*/

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %d %s %s %d\n", code, hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



__global__ void vanilla(matrix_t A, matrix_t B, matrix_t C) {
  TYPE c = 0;

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row > A.r || col > B.c) return;

  for (int i = 0; i < A.c; ++i) {
    c += A.data[ row * A.c + i] * B.data [ i * B.c + col ];
  }
  C.data[ row * C.c + col ] = c;
}

__global__ void coal() {
}

extern "C" {
  void *mult_vanilla_cuda(arg_t *args) {

    int devices;
    gpuErrchk(hipGetDeviceCount( &devices ));
    printf("%d devices\n", devices);

    hipSetDevice(1);
    struct hipDeviceProp_t properties;
    gpuErrchk( hipGetDeviceProperties( &properties, 1 ));

    printf("%s\n", properties.name);


    matrix_t A = {NULL, args->A.r, args->A.c};
    matrix_t B = {NULL, args->B.r, args->B.c};
    matrix_t C = {NULL, args->C.r, args->C.c};

    int sizeA = args->A.r * args->A.c * sizeof(TYPE);
    int sizeB = args->B.r * args->B.c * sizeof(TYPE);
    int sizeC = args->C.r * args->C.c * sizeof(TYPE);

    gpuErrchk( hipMalloc( &A.data, sizeA ));
    gpuErrchk( hipMemcpy( A.data, args->A.data, sizeA, hipMemcpyHostToDevice ));
    
    gpuErrchk( hipMalloc( &B.data, sizeB ));
    gpuErrchk( hipMemcpy( B.data, args->B.data, sizeB, hipMemcpyHostToDevice ));

    gpuErrchk( hipMalloc( &C.data, sizeC ));


    dim3 dimBlock(16,16);
    dim3 dimGrid(args->C.c, args->C.r);
    vanilla<<<dimGrid,dimBlock>>>(A,B,C);

    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(args->C.data, C.data, sizeC, hipMemcpyDeviceToHost ));

    hipFree(A.data);
    hipFree(B.data);
    hipFree(C.data);

    return NULL;
  }

  void *mult_coal_cuda(arg_t *args) {
    coal<<<1,1>>>();
    return NULL;
  }
}

